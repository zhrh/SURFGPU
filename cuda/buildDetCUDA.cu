/*
 * Copyright (C) 2009-2010 Andre Schulz, Florian Jung, Sebastian Hartte,
 *						   Daniel Trick, Christan Wojek, Konrad Schindler,
 *						   Jens Ackermann, Michael Goesele
 * Copyright (C) 2008-2009 Christopher Evans <chris.evans@irisys.co.uk>, MSc University of Bristol
 *
 * This file is part of SURFGPU.
 *
 * SURFGPU is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * SURFGPU is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with SURFGPU.  If not, see <http://www.gnu.org/licenses/>.
 */

#ifndef CUDA_BUILDDETCUDA_CU
#define CUDA_BUILDDETCUDA_CU

#ifdef __DEVICE_EMULATION__
#	include <stdio.h>
#endif

/////////////////////////////////////////////////////////
// Definitions and constants
/////////////////////////////////////////////////////////

__constant__ int dc_lobe_cache_unique[10];

__device__ float BoxIntegral(float *data, int width, int height, size_t widthStep,
							 int row, int col, int rows, int cols);

/////////////////////////////////////////////////////////
// Kernel Code (Device)
/////////////////////////////////////////////////////////

/** \brief Compute determinants
 *	\param g_img device pointer to integral image
 *	\param g_det device pointer to save resulting determinants to
 *	\param i_width integral image width
 *	\param i_height integral image height
 *	\param i_widthStep number of elements in a row of the integral image
 *	\param intervals number of intervals to compute
 *	\param o octave to compute
 *	\param step step size in X/Y direction in pixels
 *	\param border number of border pixels
 *
 *	Computation is done pixel-wise where one thread computes the determinant of
 *	one pixel.
 *
 *	Recommended execution configuration:
 *	  Thread block: { 16, 8 }
 *	  Block grid  : { ceil(i_width / block.x), ceil(i_height, block.y) }
 */
__global__ void
buildDetCUDA(float *g_img, float *g_det,
			 int i_width, int i_height, size_t i_widthStep,
			 int intervals, int o, int step, int border)
{
	// Get current interval
	const int interval_size = gridDim.x / intervals;
	// For octaves > 0, we only compute the higher 2 intervals.
	const int i = blockIdx.x / interval_size + (o > 0) * 2;

	// Get current column and row
	const int c = ((((blockIdx.x % interval_size) * blockDim.x) + threadIdx.x) * step) + border;
	const int r = ((blockIdx.y * blockDim.y + threadIdx.y) * step) + border;

	if (c >= i_width - border || r >= i_height - border)
		return;

	// Construct filter
	const int l = dc_lobe_cache_unique[o * intervals + i];
	const int w = 3 * l;
	const int b = w / 2;

	// Caluclate box integrals
	float Dxx = BoxIntegral(g_img, i_width, i_height, i_widthStep, r - l + 1, c - b, 2 * l - 1, w)
			  - BoxIntegral(g_img, i_width, i_height, i_widthStep, r - l + 1, c - l / 2, 2 * l - 1, l) * 3.0f;
	float Dyy = BoxIntegral(g_img, i_width, i_height, i_widthStep, r - b, c - l + 1, w, 2 * l - 1)
			  - BoxIntegral(g_img, i_width, i_height, i_widthStep, r - l / 2, c - l + 1, l, 2 * l - 1) * 3.0f;
	float Dxy = BoxIntegral(g_img, i_width, i_height, i_widthStep, r - l, c + 1, l, l)
			  + BoxIntegral(g_img, i_width, i_height, i_widthStep, r + 1, c - l, l, l)
			  - BoxIntegral(g_img, i_width, i_height, i_widthStep, r - l, c - l, l, l)
			  - BoxIntegral(g_img, i_width, i_height, i_widthStep, r + 1, c + 1, l, l);

	// Normalise the filter responses with respect to their size
	float inverse_area = 1.0f / (w * w);
	Dxx *= inverse_area;
	Dyy *= inverse_area;
	Dxy *= inverse_area;

	// Get the sign of the laplacian
	const float lap_sign = (Dxx + Dyy >= 0.0f ? 1.0f : -1.0f);

	// Get the determinant of hessian response
	float determinant = (Dxx * Dyy - 0.81f * Dxy * Dxy);
	unsigned int cur_intvl = o * intervals + i;
	unsigned int save_idx = cur_intvl * i_width * i_height + (r * i_width + c);
	g_det[save_idx] = determinant < 0.0f ? 0.0f : lap_sign * determinant;
}

//-------------------------------------------------------

/**	\brief Compute determinants using shared memory
 *	\param g_img device pointer to integral image
 *	\param g_det device pointer to save resulting determinants to
 *	\param i_width integral image width
 *	\param i_height integral image height
 *	\param i_widthStep number of elements in a row of the integral image
 *	\param intervals number of intervals to compute
 *	\param o octave to compute
 *	\param step step size in X/Y direction in pixels
 *	\param border number of border pixels
 *
 *	Computes the same as buildDetCUDA() but uses shared memory to cut down
 *	global memory bandwidth usage. The function uses a brute force approach for
 *	sharing the data.
 *
 *	Execution configuration:
 *	  Thread block: { 16, 4, 6 } = 384
 *	  Block grid  : { number of pixels to process in X, same in Y }
 *
 *  intervals is assumed to be 4.
 *	The kernel has only been tested for step = 2 and will very likely only
 *	compute correct results with that value.
 */
__global__ void
buildDetCUDA_smem_bf(float *g_img, float *g_det,
	int i_width, int i_height, size_t i_widthStep,
	int intervals, int o, int step, int border)
{
	__shared__ float s_data[58*34];

	// Transform thread indices from { 16, 4, z } to { 64, z }
	unsigned int t_x = threadIdx.y * blockDim.x + threadIdx.x;
	unsigned int t_y = threadIdx.z;

	unsigned int base_idx_x = blockIdx.x * 32 + t_x;
	unsigned int base_idx_y = blockIdx.y * 8 + t_y;
	unsigned int base_idx   = base_idx_y * i_widthStep
							+ base_idx_x;
	unsigned int s_base_idx = t_y * 58 + t_x;
	unsigned int s_y_offset = blockDim.z * 58;
	unsigned int g_y_offset = blockDim.z * i_widthStep;
	if (t_x < 58
		&& base_idx_x < i_width)
	{
		// Load rows 0-5
		if (base_idx_y < i_height)
		{
			s_data[s_base_idx] = g_img[base_idx];
		}
		s_base_idx += s_y_offset;
		base_idx_y += blockDim.z;
		base_idx += g_y_offset;

		// Load rows 6-11
		if (base_idx_y < i_height)
		{
			s_data[s_base_idx] = g_img[base_idx];
		}
		s_base_idx += s_y_offset;
		base_idx_y += blockDim.z;
		base_idx += g_y_offset;

		// Load rows 12-17
		if (base_idx_y < i_height)
		{
			s_data[s_base_idx] = g_img[base_idx];
		}
		s_base_idx += s_y_offset;
		base_idx_y += blockDim.z;
		base_idx += g_y_offset;

		// Load rows 18-23
		if (base_idx_y < i_height)
		{
			s_data[s_base_idx] = g_img[base_idx];
		}
		s_base_idx += s_y_offset;
		base_idx_y += blockDim.z;
		base_idx += g_y_offset;

		// Load rows 24-29
		if (base_idx_y < i_height)
		{
			s_data[s_base_idx] = g_img[base_idx];
		}
		s_base_idx += s_y_offset;
		base_idx_y += blockDim.z;
		base_idx += g_y_offset;

		// Load rows 30-33
		if (base_idx_y < i_height
			&& t_y < 4)
		{
			s_data[s_base_idx] = g_img[base_idx];
		}
	}
	__syncthreads();

	// 384 threads are used for loading data into shared memory. For computing
	// determinants only 256 threads are needed.
	if (threadIdx.z >= 4) return;

	//Construct filter
	int i = threadIdx.z;
	const int l = dc_lobe_cache_unique[o * intervals + i];
	const int w = 3 * l;
	const int b = w / 2;

	int c = threadIdx.x * step + border;
	int r = threadIdx.y * step + border;

	int gbl_c = c + blockIdx.x * blockDim.x * step;
	int gbl_r = r + blockIdx.y * blockDim.y * step;
	if (gbl_c >= i_width - border || gbl_r >= i_height - border)
		return;

	// Caluclate box integrals
	float Dxx = BoxIntegral(s_data, 58, 34, 58, r - l + 1, c - b, 2 * l - 1, w)
			  - BoxIntegral(s_data, 58, 34, 58, r - l + 1, c - l / 2, 2 * l - 1, l) * 3.0f;
	float Dyy = BoxIntegral(s_data, 58, 34, 58, r - b, c - l + 1, w, 2 * l - 1)
			  - BoxIntegral(s_data, 58, 34, 58, r - l / 2, c - l + 1, l, 2 * l - 1) * 3.0f;
	float Dxy = BoxIntegral(s_data, 58, 34, 58, r - l, c + 1, l, l)
			  + BoxIntegral(s_data, 58, 34, 58, r + 1, c - l, l, l)
			  - BoxIntegral(s_data, 58, 34, 58, r - l, c - l, l, l)
			  - BoxIntegral(s_data, 58, 34, 58, r + 1, c + 1, l, l);

	// Normalise the filter responses with respect to their size
	float inverse_area = 1.0f / (w * w);
	Dxx *= inverse_area;
	Dyy *= inverse_area;
	Dxy *= inverse_area;

	// Get the sign of the laplacian
	const float lap_sign = (Dxx + Dyy >= 0.0f ? 1.0f : -1.0f);

	// Get the determinant of hessian response
	float determinant = (Dxx * Dyy - 0.81f * Dxy * Dxy);
	unsigned int cur_intvl = o * intervals + i;
	unsigned int save_idx = cur_intvl * i_width * i_height + (gbl_r * i_width + gbl_c);
	g_det[save_idx] = determinant < 0.0f ? 0.0f : lap_sign * determinant;
}

//-------------------------------------------------------

/////////////////////////////////////////////////////////
// Device functions
/////////////////////////////////////////////////////////

//! Computes the sum of pixels within the rectangle specified by the top-left start
//! co-ordinate (row, col) and size (rows, cols).
__device__ float
BoxIntegral(float *data, int width, int height, size_t widthStep,
	int row, int col, int rows, int cols)
{
	// The subtraction by one for row/col is because row/col is inclusive.
	int r1 = min(row, height) - 1;
	int c1 = min(col, width) - 1;
	int r2 = min(row + rows, height) - 1;
	int c2 = min(col + cols, width) - 1;

	float A, B, C, D;
	A = data[r1 * widthStep + c1];
	B = data[r1 * widthStep + c2];
	C = data[r2 * widthStep + c1];
	D = data[r2 * widthStep + c2];

	return max(0.f, A - B - C + D);
}

#endif /* CUDA_BUILDDETCUDA_CU */
